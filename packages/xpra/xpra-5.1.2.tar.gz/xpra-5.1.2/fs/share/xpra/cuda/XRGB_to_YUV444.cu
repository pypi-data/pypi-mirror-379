/*
 * This file is part of Xpra.
 * Copyright (C) 2013-2021 Antoine Martin <antoine@xpra.org>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void XRGB_to_YUV444(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                             uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                             int w, int h)
{
    const uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t src_y = gy * src_h / dst_h;
    const uint32_t src_x = gx * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        uint8_t R;
        uint8_t G;
        uint8_t B;
        //one 32-bit RGB pixel at a time:
        uint32_t si = (src_y * srcPitch) + src_x * 4;
        R = srcImage[si+1];
        G = srcImage[si+2];
        B = srcImage[si+3];

        uint32_t di;
        di = (gy * dstPitch) + gx;
        dstImage[di] = __float2int_rn(0.299 * R + 0.587 * G + 0.114 * B);
        di += dstPitch*dst_h;
        dstImage[di] = __float2int_rn(-0.169 * R - 0.331 * G + 0.500 * B + 128);
        di += dstPitch*dst_h;
        dstImage[di] = __float2int_rn(0.500 * R - 0.419 * G - 0.081 * B + 128);
    }
}
